#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <libgen.h>
#include <sys/stat.h>
#include <hip/hip_runtime.h>
#include "utils/matrix_ops.cuh"
#include "utils/gpu_consts.cuh"
#include "edge_detector/edge_detectors_gpu.cuh"
#include "performance/performance_gpu.cuh"


#define BILLION 1E9
#define timeit_gpu(before, after, f, ...) {\
	hipDeviceSynchronize();\
	clock_gettime(CLOCK_MONOTONIC, &before);\
	f(__VA_ARGS__);\
	hipDeviceSynchronize();\
	clock_gettime(CLOCK_MONOTONIC, &after);\
}
#define timeit_gpu_kernel(before, after, k, ...) {\
	hipDeviceSynchronize();\
	clock_gettime(CLOCK_MONOTONIC, &before);\
	k<<<BLOCKS, THREADS>>>(__VA_ARGS__);\
	hipDeviceSynchronize();\
	clock_gettime(CLOCK_MONOTONIC, &after);\
}

void usage();
char* name(char* path);
float time_diff(struct timespec before, struct timespec after);

int main(int argc, char** argv)
{
	int* matrix = NULL;
	int* ground_truth = NULL;
	int* edge_binarized = NULL;
	int* mask = NULL;
	int* dev_matrix = NULL;
	int* dev_noisy_matrix = NULL;
	int* dev_ground_truth = NULL;
	int* dev_edge = NULL;
	int* dev_edge_binarized = NULL;
	int* dev_mask = NULL;
	hiprandState_t* states = NULL;
	FILE* fresults = NULL;
	FILE* ftimes = NULL;

	int w, h, size, steps, reps, threshold, threshold_cv, threshold_g;
	float sigma, sigma_step, sigma_min, sigma_max, similarity;
	char edge_dec, perf_fn, save_edge, dir[50], namebuffer[100], buffer[100];
	time_t t = time(NULL);
	struct tm tm = *localtime(&t);
	struct timespec tspec_before, tspec_after, tspec_tbefore, tspec_tafter;

	threshold_cv = threshold_g = 0;

	matrix = load_matrix(argv[1], &w, &h);
	ground_truth = load_matrix(argv[2], &w, &h);

	for(int i=0; i<w*h; i++)
		if(ground_truth[i] != 0 && ground_truth[i] != 1)
		{
			printf("This ground truth isn't binary. Exiting...");
			return(-1);
		}

	sigma_min = atof(argv[4]);
	sigma_max = atof(argv[5]);
	steps = atoi(argv[6]);
	reps = atoi(argv[7]);
	edge_dec = argv[8][0];
	save_edge = argv[9][0];
	perf_fn = argv[10][0];

	if(edge_dec == 'g' || edge_dec == 'a')
	{
		mask = load_mask(argv[11]);
		hipMalloc(&dev_mask, 9*sizeof(int));
		hipMemcpy(dev_mask, mask, 9*sizeof(int), hipMemcpyHostToDevice);
	}

	sprintf(dir, "%s/exec_gpu_%s_%d%02d%02d-%02d%02d%02d", argv[3], name(argv[1]), tm.tm_year + 1900, tm.tm_mon + 1, tm.tm_mday, tm.tm_hour, tm.tm_min, tm.tm_sec);
	mkdir(dir, 0755);
	if(save_edge == 'y')
	{
		sprintf(namebuffer, "%s/edges", dir);
		mkdir(namebuffer, 0755);
	}

	sprintf(namebuffer, "%s/results.dat", dir);
	fresults = fopen(namebuffer, "w");

	sprintf(namebuffer, "%s/times.dat", dir);
	ftimes = fopen(namebuffer, "w");

	size = h*w*sizeof(int);
	edge_binarized = (int*)mmalloc(h, w);
	hipMalloc(&dev_matrix, size);
	hipMalloc(&dev_noisy_matrix, size);
	hipMalloc(&dev_ground_truth, size);
	hipMalloc(&dev_edge, size);
	hipMalloc(&states, h*w*sizeof(hiprandState_t));
	if(save_edge == 'y')
		hipMalloc(&dev_edge_binarized, size);

	hipMemcpy(dev_matrix, matrix, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_ground_truth, ground_truth, size, hipMemcpyHostToDevice);


	printf("exec_gpu_%s_%d%02d%02d-%02d%02d%02d\n", name(argv[1]), tm.tm_year + 1900, tm.tm_mon + 1, tm.tm_mday, tm.tm_hour, tm.tm_min, tm.tm_sec);

	sigma_step = (sigma_max-sigma_min)/steps;
	sigma=sigma_min;
	gpu_noise_init<<<BLOCKS, THREADS>>>(time(0), states, h*w);
	for(int i=0; i<steps; i++)
	{
		for(int j=0; j<reps; j++)
		{
			sprintf(buffer, "%.3f %d", sigma, j+1);

			hipDeviceSynchronize();
			clock_gettime(CLOCK_MONOTONIC, &tspec_tbefore);

			timeit_gpu_kernel(tspec_before, tspec_after, gpu_noise_maker, states, dev_matrix, dev_noisy_matrix, 1.0, sigma, h*w);
			sprintf(buffer, "%s %.3f", buffer, 1000*time_diff(tspec_before, tspec_after));

			if(edge_dec=='c' || edge_dec=='a')
			{
				timeit_gpu_kernel(tspec_before, tspec_after, gpu_edge_detector_cv, dev_noisy_matrix, dev_edge, w, h);
				sprintf(buffer, "%s %.3f", buffer, 1000*time_diff(tspec_before, tspec_after));

				if(perf_fn=='o' || perf_fn=='a')
				{
					timeit_gpu(tspec_before, tspec_after, gpu_find_threshold_optimized, 0, threshold_cv, 8, 2, 0.5, dev_edge, dev_ground_truth, w, h, gpu_edge_comparison, &threshold_cv, &similarity);
					sprintf(buffer, "%s %.3f", buffer, 1000*time_diff(tspec_before, tspec_after));

					fprintf(fresults, "%.3f %d cv opt %d %.6f\n", sigma, j+1, threshold_cv, similarity);

					sprintf(namebuffer, "%s/edges/%.3f_%d_cv_opt.dat", dir, sigma, j+1);
					if(save_edge == 'y')
					{
						gpu_binarization<<<BLOCKS, THREADS>>>(dev_edge, dev_edge_binarized, w, h, threshold_cv);
						hipMemcpy(edge_binarized, dev_edge_binarized, size, hipMemcpyDeviceToHost);
						save_matrix(namebuffer, edge_binarized, w, h);
					}
				}
				if(perf_fn=='e' || perf_fn=='a')
				{
					timeit_gpu(tspec_before, tspec_after, gpu_find_threshold_exhaustive, dev_edge, dev_ground_truth, w, h, gpu_edge_comparison, &threshold, &similarity);
					sprintf(buffer, "%s %.3f", buffer, 1000*time_diff(tspec_before, tspec_after));

					fprintf(fresults, "%.3f %d cv exh %d %.6f\n", sigma, j+1, threshold, similarity);

					sprintf(namebuffer, "%s/edges/%.3f_%d_cv_exh.dat", dir, sigma, j+1);
					if(save_edge == 'y')
					{
						gpu_binarization<<<BLOCKS, THREADS>>>(dev_edge, dev_edge_binarized, w, h, threshold);
						hipMemcpy(edge_binarized, dev_edge_binarized, size, hipMemcpyDeviceToHost);
						save_matrix(namebuffer, edge_binarized, w, h);
					}
				}
			}
			if(edge_dec=='g' || edge_dec=='a')
			{
				timeit_gpu_kernel(tspec_before, tspec_after, gpu_edge_detector_g, dev_noisy_matrix, dev_edge, w, h, dev_mask);
				sprintf(buffer, "%s %.3f", buffer, 1000*time_diff(tspec_before, tspec_after));

				if(perf_fn=='o' || perf_fn=='a')
				{
					timeit_gpu(tspec_before, tspec_after, gpu_find_threshold_optimized, 0, threshold_g, 8, 2, 0.5, dev_edge, dev_ground_truth, w, h, gpu_edge_comparison, &threshold_g, &similarity);
					sprintf(buffer, "%s %.3f", buffer, 1000*time_diff(tspec_before, tspec_after));

					fprintf(fresults, "%.3f %d g opt %d %.6f\n", sigma, j+1, threshold_g, similarity);

					sprintf(namebuffer, "%s/edges/%.3f_%d_g_opt.dat", dir, sigma, j+1);
					if(save_edge == 'y')
					{
						gpu_binarization<<<BLOCKS, THREADS>>>(dev_edge, dev_edge_binarized, w, h, threshold_g);
						hipMemcpy(edge_binarized, dev_edge_binarized, size, hipMemcpyDeviceToHost);
						save_matrix(namebuffer, edge_binarized, w, h);
					}
				}
				if(perf_fn=='e' || perf_fn=='a')
				{
					timeit_gpu(tspec_before, tspec_after, gpu_find_threshold_exhaustive, dev_edge, dev_ground_truth, w, h, gpu_edge_comparison, &threshold, &similarity);
					sprintf(buffer, "%s %.3f", buffer, 1000*time_diff(tspec_before, tspec_after));

					fprintf(fresults, "%.3f %d g exh %d %.6f\n", sigma, j+1, threshold, similarity);

					sprintf(namebuffer, "%s/edges/%.3f_%d_g_exh.dat", dir, sigma, j+1);
					if(save_edge == 'y')
					{
						gpu_binarization<<<BLOCKS, THREADS>>>(dev_edge, dev_edge_binarized, w, h, threshold);
						hipMemcpy(edge_binarized, dev_edge_binarized, size, hipMemcpyDeviceToHost);
						save_matrix(namebuffer, edge_binarized, w, h);
					}
				}
			}
			hipDeviceSynchronize();
			clock_gettime(CLOCK_MONOTONIC, &tspec_tafter);
			sprintf(buffer, "%s %.3f", buffer, 1000*time_diff(tspec_tbefore, tspec_tafter));

			printf("%s\n", buffer);
			fprintf(ftimes, "%s\n", buffer);
		}
		sigma += sigma_step;
	}

	fclose(fresults);
	fclose(ftimes);

	mfree(matrix);
	mfree(ground_truth);
	mfree(edge_binarized);
	mfree(mask);
	hipFree(dev_matrix);
	hipFree(dev_noisy_matrix);
	hipFree(dev_ground_truth);
	hipFree(dev_edge);
	hipFree(dev_edge_binarized);
	hipFree(dev_mask);


	hipDeviceReset();
	return 0;
}

void usage()
{
	printf(
		"Usage:\n"
		);
}

char* name(char* path)
{
    int i;
    char* base;

    base = basename(path);

    for(i=0;base[i]!='.' && base[i]!='\0';i++);
    base[i] = '\0';

    return base;
}

float time_diff(struct timespec before, struct timespec after)
{
	return (after.tv_sec - before.tv_sec) + (after.tv_nsec - before.tv_nsec) / BILLION;
}

/*
	exec:
	./main <matrix_file> <ground_truth_file> <output_dir> <sigma_min> <sigma_max> <steps> <reps> <c/g/a> <save_edge(y/n)> <o/e/a> <mask_file>
	out:
	(results) <sigma> <rep> <edge_detector> <perf_fn> <threshold> <similarity>
	(times) <sigma> <rep> <noise_maker> <edge_detector> <perf_fn> ... <+edges> ...

	Parámetros:
	- Matriz de imagen original
	- Matrix de terreno de la verdad (la matriz debe ser de 0 y 1 sólamente)
	- Directorio donde se guardarán los archivos(tiempos, resultados y mejores contornos)
	- Parámetros de ruido(sigma_min, sigma_max, intervalos, repeticiones)
	- Detectores de ruido a usar(cv, g). Si g agregar máscara como último parámetro.
	- Funciones de performance a usar(o, e)

	Salida:
	- Mejor contorno por cada detector, por cada función de performance y por cada matriz de ruido generada (<=4 por matriz de ruido).
	- Archivo con tiempos para generación de ruido, detectores de contorno y performance
	- Tiempos totales


	Procedimiento:
	- Cargar matriz original y terreno de la verdad
	por cada valor de ruido:
		por cada repetición para valor de ruido:
			- Generar matriz ruidosa
			por cada detector seleccionado:
				- Obtener contorno
				por cada función de performance:
					- Comparar contorno con terreno de la verdad
					- Guardar a archivo el mejor contorno
			- Imprimir resultados y tiempos a archivos separados
			- Guardar matriz ruidosa
	- Sumar tiempos e imprimir a archivo(o stdout)

	TODO:
	Arreglar la fuga de memoria!
	
	guardar:
	- (mejores contornos) nombreimg_sigma_rep_detector_perf.dat
	- (tiempos)
*/